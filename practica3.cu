#include "hip/hip_runtime.h"
#include "omp.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

#define BLOCKSPERGRID 1

__global__ void kern_mat_mul(const unsigned char *A, const int *K, unsigned char *B, int A_size, int max_row, int K_size, int channels){
    
    __shared__ int kernel[9][9];

    for(int i=0; i<K_size*K_size; i++){
        kernel[i/K_size][i-(i/K_size)*K_size] = *(K + i);
    }

    int row = blockDim.x * blockIdx.x + threadIdx.x;

    int ai = row * A_size * channels;

    if(row <= max_row) {
        for(int c=0; c<(A_size-K_size)*channels; c++, ai++) {
            int sum = 0; 
            for (int i=0; i<K_size; i++, ai+=(A_size-K_size)*channels) { // Iteracion sobre la matriz
                for (int j=0; j<K_size; j++, ai+=channels) {
                    sum += ((*(A+ai))*(kernel[i][j]));
                }
            }
            
            if (sum > 255) sum = 255;
            else if (sum <0) sum = 0;
            *(B + row * (A_size-K_size) * channels + c) = (uint8_t) sum;
        }
    }
}

int main(int argc, char **argv){
    hipError_t err = hipSuccess;

    if(argc < 4){ // Verficacion de los argumentos necesarios
        printf("Debe ingresar la ruta de la imagen, el numero de kernel (1-3), la opcion de filtro (c (solo contraste en color), g (solo contraste en escala de grises) ó cg (ambas opciones))) y el número de hilos");
        return 0;
    }
      
    int width, height, channels;
    int k = argv[2][0] - '0' - 1; // Kernel deseado

    if (k>3 || k<0) { // Verificación de k valido
        printf("Kernel invalido\n");
        return 0;
    }

    int threads_per_block = atoi(argv[4]);
    
    unsigned char *img = stbi_load(argv[1], &width, &height, &channels, 0); // Cargue de la imagen
    if(img == NULL) { //Verificacion de la imagen
       printf("Error cargando la imagen\n");
       exit(1);
    }

    const int k_size = k == 0 ? 9 : 3;
    
    int kernels[][9][9] = {{ // Kernels
        { 0, 0, 0,-1,-1,-1, 0, 0, 0},
        { 0,-2,-3,-3,-3,-3,-3,-2, 0},
        { 0,-3,-2,-1,-1,-1,-2,-3, 0},
        {-1,-3,-1, 9, 9, 9,-1,-3,-1},
        {-1,-3,-1, 9,19, 9,-1,-3,-1},
        {-1,-3,-1, 9, 9, 9,-1,-3,-1},
        { 0,-3,-2,-1,-1,-1,-2,-3, 0},
        { 0,-2,-3,-3,-3,-3,-3,-2, 0},
        { 0, 0, 0,-1,-1,-1, 0, 0, 0},
    },{ {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    },{ { 1, 2, 1},
        { 0, 0, 0},
        {-1,-2,-1}
    }};

    struct timeval tval_before, tval_after; //Declaracion de variables de tiempo
    gettimeofday(&tval_before, NULL);
    
    int width_t = width - k_size; //Dimensiones de la nueva imagen
    int height_t = height - k_size;
    size_t cont_img_size = width_t * height_t * channels;

    //
    
    unsigned char *cont_mult_img = (unsigned char *) malloc(cont_img_size);// Reserva de memoria para la nueva imagen
    if(cont_mult_img == NULL){ printf("Error al reservar memoria img host\n"); return(1);}

    //

    int* h_k = (int*) malloc(k_size*k_size*sizeof(int));
    if(h_k == NULL){ printf("Error al reservar memoria kernel host\n"); return(1);}
    
    for(int i=0; i<k_size*k_size; i++){
        *(h_k+i) = kernels[k][i/k_size][i-(i/k_size)*k_size];
    }

    int* d_k = NULL;

    err = hipMalloc((void**)&d_k, k_size*k_size*sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device kernel matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // 

    unsigned char *d_img = NULL;

    err = hipMalloc((void**)&d_img, width*height*channels);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device img matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    unsigned char *d_cont_img = NULL;

    err = hipMalloc((void**)&d_cont_img, cont_img_size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device cont img matrix (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    err = hipMemcpy(d_k, h_k, k_size*k_size*sizeof(int), hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy kernel matrix to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    err = hipMemcpy(d_img, img, width*height*channels, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy img matrix to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    int n_threads = height_t;
    int blocks_per_grid = n_threads/threads_per_block+1;

    printf("CUDA kernel launch with %d blocks of %d threads\n", blocks_per_grid, threads_per_block);
    kern_mat_mul<<<blocks_per_grid, threads_per_block>>>(d_img, d_k, d_cont_img, width, height, k_size, channels);
    
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    err = hipMemcpy(cont_mult_img, d_cont_img, cont_img_size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //

    err = hipFree(d_k);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free kernel matrix in device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_img);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free img matrix in device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    } 

    err = hipFree(d_cont_img);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free cont img matrix in device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    } 

    stbi_write_jpg("img/img_contraste_color.jpg", width_t, height_t, channels, cont_mult_img, 100); // Guardado de la nueva imagen
    stbi_image_free(cont_mult_img);

    gettimeofday(&tval_after, NULL); // Medicion de tiempo
    printf("Tiempo de procesamiento de %s con kernel=%s opcion=%s y %d hilos por block: %f\n", argv[1],argv[2],argv[3], threads_per_block, (tval_after.tv_sec + tval_after.tv_usec/1000000.0) - (tval_before.tv_sec + tval_before.tv_usec/1000000.0));
}